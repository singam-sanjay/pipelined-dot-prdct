
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include"hipblas.h"

using namespace std;
typedef float TYPE;

int main()
{
	size_t N;
	cout << "N:";cin >> N;

	TYPE *arr,*d_arr;

	try
	{
		arr = new TYPE[N];
	}
	catch(exception e)
	{
		cerr << "Unable to allocate mem for arr: " << e.what() << '\n';
		return EXIT_FAILURE;
	}

	if( hipMalloc( &d_arr,sizeof(TYPE)*N )!=hipSuccess )
	{
		cerr << "Unable to allocate d_arr :" << hipGetErrorString( hipGetLastError() ) << '\n';
		return EXIT_FAILURE;
	}

	if( hipFree( d_arr )!=hipSuccess )
	{
		cerr << "Unable to deallocate mem for d_arr: " << hipGetErrorString( hipGetLastError() ) << '\n';
		return EXIT_FAILURE;
	}

	try
	{
		delete [] arr;
	}
	catch(exception e)
	{
		cerr << "Unable to deallocate mem for arr: " << e.what() << '\n';
		return EXIT_FAILURE;
	}

	return EXIT_SUCCESS;
}
