/* kernels and cuBLAS call */
__constant__ TYPE alpha = -1.0, beta = 1.0;
TYPE *gpu_addr_alpha,*gpu_addr_beta;

const char* __cuBLAS_error_string( hipblasStatus_t stat )
{
  switch( stat )
  {
    case HIPBLAS_STATUS_SUCCESS          : return "the operation completed successfully";
    case HIPBLAS_STATUS_NOT_INITIALIZED  : return "the library was not initialized";
    case HIPBLAS_STATUS_ARCH_MISMATCH    : return "the device does not support double-precision";
    case HIPBLAS_STATUS_EXECUTION_FAILED : return "HIPBLAS_STATUS_EXECUTION_FAILED";
    default                             : return "<Unknown Error>";
  }
}

void setup_cuBLAS_func_env()
{
	hipError_t err;
	err = hipGetSymbolAddress( (void**)&gpu_addr_alpha , alpha );
	if( err!=hipSuccess )
	{
		err_sstr << "Error while getting GPU address of alpha::" << hipGetErrorString(err) << '\n';
		throw_str_excptn();
	}
	err = hipGetSymbolAddress( (void**)&gpu_addr_beta , beta );
	if( err!=hipSuccess )
	{
		err_sstr << "Error while getting GPU address of beta::" << hipGetErrorString(err) << '\n';
		throw_str_excptn();
	}
	hipblasStatus_t stat;
	stat = hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	if( stat != HIPBLAS_STATUS_SUCCESS )
	{
		err_sstr << "hipblasSetPointerMode::" << __cuBLAS_error_string(stat) << ".\n";
		throw_str_excptn();
	}
}

void seq()
{
  hipblasStatus_t stat;
  for( int row_num = 0 ; row_num<k ; ++row_num )
  {
    stat = hipblasDaxpy( handle, N, gpu_addr_alpha, gpu_vec, 1, gpu_wrk_mat+N*row_num, 1 );
    if( stat!=HIPBLAS_STATUS_SUCCESS )
    {
      err_sstr << __func__ << "::hipblasDaxpy::" << __cuBLAS_error_string(stat);
      throw_str_excptn();
    }
    stat = hipblasDnrm2( handle, N, gpu_wrk_mat+N*row_num, 1, gpu_res+row_num );
    if( stat!=HIPBLAS_STATUS_SUCCESS )
    {
      err_sstr << __func__ << "::hipblasDnrm2::" << __cuBLAS_error_string(stat);
      throw_str_excptn();
    }
  }
}

void par_OpenMP()
{
  hipblasStatus_t stat;
  #pragma omp parallel for private(stat)
  for( int row_num = 0 ; row_num<k ; ++row_num )
  {
    stat = hipblasDaxpy( handle, N, gpu_addr_alpha, gpu_vec, 1, gpu_wrk_mat+N*row_num, 1 );
    if( stat!=HIPBLAS_STATUS_SUCCESS )
    {
      err_sstr << __func__ << "::hipblasDaxpy::" << __cuBLAS_error_string(stat);
      throw_str_excptn();
    }
    stat = hipblasDnrm2( handle, N, gpu_wrk_mat+N*row_num, 1, gpu_res+row_num );
    if( stat!=HIPBLAS_STATUS_SUCCESS )
    {
      err_sstr << __func__ << "::hipblasDnrm2::" << __cuBLAS_error_string(stat);
      throw_str_excptn();
    }
  }
}

__device__ void dyn_par_kernel( hipblasHandle_t handle, int N, TYPE *gpu_addr_alpha, TYPE *gpu_vec, TYPE *gpu_wrk_mat, TYPE *gpu_res )
{
  int row_num = blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;
  if( row_num>=N )
  {
    return;
  }
  hipblasStatus_t stat;
  stat = hipblasDaxpy( handle, N, gpu_addr_alpha, gpu_vec, 1, gpu_wrk_mat+N*row_num, 1 );
  if( stat!=HIPBLAS_STATUS_SUCCESS )
  {
    // Globally visible data / Write error message to string in global memory
    printf("%s::%i::hipblasDaxpy::error",__func__,row_num);
    asm("trap;");
  }
  stat = hipblasDnrm2( handle, N, gpu_wrk_mat+N*row_num, 1, gpu_res+row_num );
  if( stat!=HIPBLAS_STATUS_SUCCESS )
  {
    // Globally visible data / Write error message to string in global memory
    printf("%s::%i::hipblasDnrm2::error",__func__,row_num);
    asm("trap;");
  }
}

void par_dyn_parll()
{
  #define NUMBER_OF_BLOCKS ( (k-1)/THREADS_PER_BLOCK + 1 )
  dyn_par_kernel<<<NUMBER_OF_BLOCKS,THREADS_PER_BLOCK>>>(handle,N,gpu_addr_alpha,gpu_wrk_mat,gpu_res);
  hipError_t err = hipGetLastError();
  if( err!=hipSuccess )
  {
    err_sstr << __func__ << hipGetErrorString(err);
    throw_str_excptn();
  }
}
