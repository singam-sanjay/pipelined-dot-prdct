/* kernels and cuBLAS call */
__constant__ TYPE alpha = -1.0, beta = 1.0;
TYPE *gpu_addr_alpha,*gpu_addr_beta;

const char* __cuBLAS_error_string( hipblasStatus_t stat )
{
  switch( stat )
  {
    case HIPBLAS_STATUS_SUCCESS          : return "the operation completed successfully";
    case HIPBLAS_STATUS_NOT_INITIALIZED  : return "the library was not initialized";
    case HIPBLAS_STATUS_ARCH_MISMATCH    : return "the device does not support double-precision";
    case HIPBLAS_STATUS_EXECUTION_FAILED : return "HIPBLAS_STATUS_EXECUTION_FAILED";
    default                             : return "<Unknown Error>";
  }
}

void setup_cuBLAS_func_env()
{
	hipError_t err;
	err = hipGetSymbolAddress( (void**)&gpu_addr_alpha , alpha );
	if( err!=hipSuccess )
	{
		err_sstr << "Error while getting GPU address of alpha::" << hipGetErrorString(err) << '\n';
		throw_str_excptn();
	}
	err = hipGetSymbolAddress( (void**)&gpu_addr_beta , beta );
	if( err!=hipSuccess )
	{
		err_sstr << "Error while getting GPU address of beta::" << hipGetErrorString(err) << '\n';
		throw_str_excptn();
	}
	hipblasStatus_t stat;
	stat = hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	if( stat != HIPBLAS_STATUS_SUCCESS )
	{
		err_sstr << "hipblasSetPointerMode::" << __cuBLAS_error_string(stat) << ".\n";
		throw_str_excptn();
	}
}

void seq()
{
  hipblasStatus_t stat;
  for( int row_num = 0 ; row_num<k ; ++row_num )
  {
    stat = hipblasDaxpy( handle, N, gpu_addr_alpha, gpu_vec, 1, gpu_wrk_mat+N*row_num, 1 );
    if( stat!=HIPBLAS_STATUS_SUCCESS )
    {
      err_sstr << __func__ << "::hipblasDaxpy::" << __cuBLAS_error_string(stat);
      throw_str_excptn();
    }
    stat = hipblasDnrm2( handle, N, gpu_wrk_mat+N*row_num, 1, gpu_res+row_num );
    if( stat!=HIPBLAS_STATUS_SUCCESS )
    {
      err_sstr << __func__ << "::hipblasDnrm2::" << __cuBLAS_error_string(stat);
      throw_str_excptn();
    }
  }
}
