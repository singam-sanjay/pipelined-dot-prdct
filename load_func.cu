/* Load and Write Functions */
#include <fstream>

void ld_data_frm_binfile( const char *file_name, size_t offset, const char *var_name, TYPE *ptr, size_t num, int line )
{
  try
  {
    std::ifstream f( file_name, std::ifstream::binary );
    if( !f.is_open() )
    {
      err_sstr << "ld__frm_file_to_CPU::Unable to open " << file_name << '\n';
      throw_str_excptn();
    }
    f.exceptions( std::ifstream::failbit | std::ifstream::badbit | std::ifstream::eofbit );
    f.seekg( offset, std::ios_base::beg );
    f.read( (char*)ptr, sizeof(TYPE)*num );
    f.close();
  }
  catch( std::ios_base::failure &fail )
  {
    err_sstr << "ld__frm_file_to_CPU::" << line << "::failure::" << file_name << "::" << fail.what() << '\n';
    throw_str_excptn();
  }
}

void ld__frm_file_to_CPU( const char * dataset, const char * computed_result = NULL )
{
  #define ld_cpu_MACRO(file,offset,var,size) { ld_data_frm_binfile(file,offset,#var,var,size,__LINE__); }
  ld_cpu_MACRO(dataset, 0,cpu_vec,1*N);
  ld_cpu_MACRO(dataset,+N,cpu_mat,k*N);
  #ifdef DEBUG
  ld_cpu_MACRO(computed_result,0,cpu_res,k*N);
  #endif
  #undef ld_cpu_MACRO
}

#include "hipblas.h"

void ld_GPU_vec( TYPE* d_vec, TYPE* vec, size_t bytes, const char* var_name, const char* d_var_name )
{
	hipblasStatus_t stat;
	stat = hipblasSetVector( bytes,sizeof(TYPE),vec,1,d_vec,1 );
	if( stat!=HIPBLAS_STATUS_SUCCESS )
	{
		err_sstr << "ld__frm_CPU_to_GPU::" << var_name << "->" << d_var_name << "::";
    switch( stat )
    {
      case HIPBLAS_STATUS_NOT_INITIALIZED: err_sstr << "The library was not initialized.\n";break;
      case HIPBLAS_STATUS_INVALID_VALUE: err_sstr << "The parameters incx , incy , elemSize<=0\n";break;
      case HIPBLAS_STATUS_MAPPING_ERROR: err_sstr << "There was an error accessing GPU memory\n";break;
      default:			  err_sstr << "Got something else\n";break;
    }
    throw_str_excptn();
	}
	else
	{
		return;
	}
}

void ld_GPU_mat( TYPE* d_mat, TYPE* mat, size_t rows, size_t cols, const char* var_name, const char* d_var_name )
{
	hipblasStatus_t stat;
	stat = hipblasSetMatrix( rows,cols,sizeof(TYPE),mat,rows,d_mat,rows );
	if( stat!=HIPBLAS_STATUS_SUCCESS )
	{
		err_sstr << "ld__frm_CPU_to_GPU::" << var_name << "->" << d_var_name << "::";
    switch( stat )
    {
      case HIPBLAS_STATUS_NOT_INITIALIZED: err_sstr << "The library was not initialized.\n";break;
      case HIPBLAS_STATUS_INVALID_VALUE: err_sstr << "The parameters rows, cols<0 or elemSize, lda, ldb<=0\n";break;
      case HIPBLAS_STATUS_MAPPING_ERROR: err_sstr << "There was an error accessing GPU memory\n";break;
      default:			  err_sstr << "Got something else\n";break;
    }
    throw_str_excptn();
	}
	else
	{
		return;
	}
}

void ld__frm_CPU_to_GPU()
{
  #define ld_GPU_vec_MACRO( d_var,var,bytes ) { ld_GPU_vec( d_var,var,bytes,#var,#d_var ); }
  #define ld_GPU_mat_MACRO( d_var,var,rows,cols ) { ld_GPU_mat( d_var,var,rows,cols,#var,#d_var ); }
  ld_GPU_vec_MACRO( gpu_vec,cpu_vec,N );
  ld_GPU_mat_MACRO( gpu_rep_mat,cpu_mat,k,N );
  #undef ld_GPU_mat_MACRO
  #undef ld_GPU_mat_MACRO
}

void rp__frm_rplca_to_wrkspc_on_GPU()
{
  hipError_t status;
  status = hipMemcpy(gpu_wrk_mat,gpu_rep_mat,k*N,hipMemcpyDeviceToDevice);
  if( status==hipSuccess )
  {
    return;
  }
  err_sstr << __func__ << "::" ;
  switch( status )
  {
    case hipErrorInvalidValue : err_sstr << "parameters passed to the API call is not within an acceptable range of values.\n";break;
    case hipErrorInvalidDevicePointer : err_sstr << "at least one device pointer passed to the API call is not a valid device pointer\n";break;
    case hipErrorInvalidMemcpyDirection 	 : err_sstr << "direction of the memcpy passed to the API call is not one of the types specified by hipMemcpyKind\n";break;
  }
  throw_str_excptn();
}

#ifdef DEBUG
void wb__to_CPU_frm_GPU()
{
  hipblasStatus_t status;
  status = hipblasGetVector( size_vec,sizeof(TYPE),gpu_res,1,cpu_res,1 );
  if( status!=HIPBLAS_STATUS_SUCCESS )
  {
    err_sstr << __func__ << "::gpu_res->cpu_res::";
    switch( status )
    {
      case HIPBLAS_STATUS_NOT_INITIALIZED: err_sstr << "The library was not initialized.\n";break;
      case HIPBLAS_STATUS_INVALID_VALUE: err_sstr << "The parameters incx , incy , elemSize<=0\n";break;
      case HIPBLAS_STATUS_MAPPING_ERROR: err_sstr << "There was an error accessing GPU memory\n";break;
      default:			  err_sstr << "Got something else\n";break;
    }
    throw_str_excptn();
  }
}

void wb__to_file_frm_CPU()
{
  /*
    Not writing any supporing function for this since only 1 variable has to written back
    */
  try
  {
    ofstream f( file_name, std::ifstream::binary );
    if( !f.is_open() )
    {
      err_sstr << __func__ << "::Unable to open " << file_name << '\n';
      throw_str_excptn();
    }
    f.exceptions( std::ifstream::failbit | std::ifstream::badbit | std::ifstream::eofbit );
    f.write( ptr, sizeof(TYPE)*num );
    f.close();
  }
  catch( std::ios_base::failure &fail )
  {
    err_sstr << __func__ << "::failure::" << file_name << "::" << fail.what() << '\n';
    throw_str_excptn();
  }
}
#endif
